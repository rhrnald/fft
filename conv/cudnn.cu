#include "convolution.hpp"
#include "gpuTimer.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdio.h>

#define checkCuda(expr)                                                        \
  do {                                                                         \
    hipError_t err = (expr);                                                  \
    if (err != hipSuccess) {                                                  \
      printf("CUDA Error: %s\n", hipGetErrorString(err));                     \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

#define checkCudnn(expr)                                                       \
  do {                                                                         \
    hipdnnStatus_t err = (expr);                                                \
    if (err != HIPDNN_STATUS_SUCCESS) {                                         \
      printf("cuDNN Error: %s\n", hipdnnGetErrorString(err));                   \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

void convolution_cudnn(float *h_input, float *h_filter, float *h_output, int N,
                       int f) {
  hipdnnHandle_t handle;
  checkCudnn(hipdnnCreate(&handle));

  hipdnnTensorDescriptor_t in_desc, out_desc;
  hipdnnFilterDescriptor_t filter_desc;
  hipdnnConvolutionDescriptor_t conv_desc;

  int out_size = N - f + 1;

  checkCudnn(hipdnnCreateTensorDescriptor(&in_desc));
  checkCudnn(hipdnnCreateTensorDescriptor(&out_desc));
  checkCudnn(hipdnnCreateFilterDescriptor(&filter_desc));
  checkCudnn(hipdnnCreateConvolutionDescriptor(&conv_desc));

  checkCudnn(hipdnnSetTensor4dDescriptor(in_desc, HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT, 1, 1, N, N));
  checkCudnn(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT,
                                        HIPDNN_TENSOR_NCHW, 1, 1, f, f));
  checkCudnn(hipdnnSetConvolution2dDescriptor(
      conv_desc, 0, 0, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

  int n, c, h, w;
  checkCudnn(hipdnnGetConvolution2dForwardOutputDim(
      conv_desc, in_desc, filter_desc, &n, &c, &h, &w));
  checkCudnn(hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT, n, c, h, w));

  float *d_input, *d_filter, *d_output;
  checkCuda(hipMalloc(&d_input, sizeof(float) * N * N));
  checkCuda(hipMalloc(&d_filter, sizeof(float) * f * f));
  checkCuda(hipMalloc(&d_output, sizeof(float) * h * w));

  checkCuda(hipMemcpy(d_input, h_input, sizeof(float) * N * N,
                       hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_filter, h_filter, sizeof(float) * f * f,
                       hipMemcpyHostToDevice));

  hipdnnConvolutionFwdAlgoPerf_t perf_results[1];
  int returned_algo_count = 0;
  checkCudnn(hipdnnFindConvolutionForwardAlgorithm(
      handle, in_desc, filter_desc, conv_desc, out_desc, 1,
      &returned_algo_count, perf_results));

  hipdnnConvolutionFwdAlgo_t algo = perf_results[0].algo;

  size_t workspace_bytes = 0;
  checkCudnn(hipdnnGetConvolutionForwardWorkspaceSize(
      handle, in_desc, filter_desc, conv_desc, out_desc, algo,
      &workspace_bytes));
  void *workspace = nullptr;
  checkCuda(hipMalloc(&workspace, workspace_bytes));

  float alpha = 1.0f, beta = 0.0f;

  GpuTimer timer;
  timer.Start();
  checkCudnn(hipdnnConvolutionForward(
      handle, &alpha, in_desc, d_input, filter_desc, d_filter, conv_desc, algo,
      workspace, workspace_bytes, &beta, out_desc, d_output));
  timer.Stop();

  checkCuda(hipMemcpy(h_output, d_output, sizeof(float) * h * w,
                       hipMemcpyDeviceToHost));

  long long ops = 2LL * out_size * out_size * f * f;
  float time_ms = timer.Elapsed();
  float gflops = ops / (time_ms * 1e6f);
  printf("[cuDNN] Time: %.3f ms, GFLOPS: %.2f\n", time_ms, gflops);

  hipFree(d_input);
  hipFree(d_filter);
  hipFree(d_output);
  hipFree(workspace);
  hipdnnDestroyTensorDescriptor(in_desc);
  hipdnnDestroyTensorDescriptor(out_desc);
  hipdnnDestroyFilterDescriptor(filter_desc);
  hipdnnDestroyConvolutionDescriptor(conv_desc);
  hipdnnDestroy(handle);
}