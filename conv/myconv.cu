#include "hip/hip_runtime.h"
#include "gpuTimer.h"
#include "my_fft.h"

#include <hipfft/hipfft.h>
#include <cufftdx.hpp>
#include <typeinfo>

#define DEBUG_VAR(x) std::cout << #x << ": " << (x) << std::endl;

#define checkCuda(expr)                                                        \
  do {                                                                         \
    hipError_t err = (expr);                                                  \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,         \
              hipGetErrorString(err));                                        \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

using namespace cufftdx;

static constexpr unsigned int tile_size = 64;

__launch_bounds__(128)
__global__ void my_conv_kernel(float *d_input,
                            cufftdx::complex<float> *d_filter,
                            float *d_output, int input_size, int f, const hipFloatComplex* __restrict__ W_64) {
  using complex_type = typename cufftdx::complex<float>;

  // Allocate register
  float2 thread_data[32];
  __shared__ __align__(alignof(float4)) float2 tile[tile_size*(tile_size/2+1)];

  // thread idx
  const int lane_id = threadIdx.x;
  const int warp_id = threadIdx.y;

  const int output_size = input_size - f + 1;
  const int valid_tile_size = tile_size - f + 1;

  // gmem -> smem
  for (int local_row = warp_id; local_row < tile_size; local_row+=blockDim.y) {
    int global_row = local_row + blockIdx.y * valid_tile_size;

    int local_col = lane_id;
    int global_col = local_col*2+blockIdx.x * valid_tile_size;

    if(global_row < input_size && global_col < input_size) {
      tile[local_col + (tile_size / 2 + 1) * local_row] = *reinterpret_cast<float2 *>(d_input+global_row * input_size + global_col);
    } else {
      tile[local_col + (tile_size / 2 + 1) * local_row] = make_float2(0.0f, 0.0f);
    }
  }


  __syncthreads();

  // row-wise fft
  for(int i=0; i<32; i++) {
    int row = lane_id/4 + (i/16)*8 + warp_id*16;
    int col = lane_id%4 + (i%16)*4;
    thread_data[i].x = *(((float*)tile) +row*(tile_size + 2)+reverse_2bit_groups<6>(col));
    thread_data[i].y = 0.0f;
  }

  fft_kernel_r64_b16<false>((hipFloatComplex *)thread_data, W_64);

  for(int i=0; i<32; i++) {
    int row = (lane_id/4) + (i/16)*8 + warp_id*16;
    int col = (lane_id%4) * 16 + (i%16);
    if(col<tile_size/2+1) {
      tile[row * (tile_size/2+1) + col] = thread_data[i];
    }
  }
  __syncthreads();

  // col-wise fft
  for(int i=0; i<32; i++) {
    int col = lane_id/4 + (i/16)*8 + warp_id*16;
    int row = lane_id%4 + (i%16)*4;
    if(col<tile_size/2+1) {
      thread_data[i] = tile[reverse_2bit_groups<6>(row) * (tile_size/2+1) + col];
    }
  }

  if(warp_id<3) fft_kernel_r64_b16<false>((hipFloatComplex *)thread_data, W_64);

  for(int i=0; i<32; i++) {
    int col = (lane_id/4) + (i/16)*8 + warp_id*16;
    int row = (lane_id%4) * 16 + (i%16);
    if(col<tile_size/2+1) {
      tile[row * (tile_size/2+1) + col] = thread_data[i];
    }
  }
  __syncthreads();

  //element-wise mult
  for(int row=lane_id; row<tile_size; row+=32) {
    for (int col = warp_id; col <=tile_size/2; col+=blockDim.y) {
      ((complex_type*)tile)[row * (tile_size/2+1)  + col] *= d_filter[row * (tile_size/2+1)  + col];
    }
  }

  __syncthreads();
  // col-wise ifft
  for(int i=0; i<32; i++) {
    int col = lane_id/4 + (i/16)*8 + warp_id*16;
    int row = lane_id%4 + (i%16)*4;
    if(col<tile_size/2+1) {
      thread_data[i] = tile[reverse_2bit_groups<6>(row) * (tile_size/2+1) + col];
    }
  }

  if(warp_id<3) fft_kernel_r64_b16<true>((hipFloatComplex *)thread_data, W_64);

  for(int i=0; i<32; i++) {
    int col = (lane_id/4) + (i/16)*8 + warp_id*16;
    int row = (lane_id%4) * 16 + (i%16);
    if(col<tile_size/2+1) {
      tile[row * (tile_size/2+1) + col] = thread_data[i];
    }
  }

  __syncthreads();
  // if(local_thread_id==0 && local_fft_id==0) {
  //   for(int i=0 ; i<tile_size;i++) {
  //     for (int j = 0; j <=tile_size/2; j++) {
  //       printf("%f %f ", tile[i*(tile_size/2+1)+j].x,
  //       tile[i*(tile_size/2+1)+j].y);
  //     }
  //     printf("\n");
  //   }
  //   printf("\n\n");
  // }
  // row-wise ifft
  for(int i=0; i<32; i++) {
    int row = lane_id/4 + (i/16)*8 + warp_id*16;
    int col = lane_id%4 + (i%16)*4;
    if(reverse_2bit_groups<6>(col)<tile_size/2+1) {
      thread_data[i] = tile[row * (tile_size/2+1) + reverse_2bit_groups<6>(col)];
    } else {
      thread_data[i] = tile[row * (tile_size/2+1) + tile_size-reverse_2bit_groups<6>(col)];
      thread_data[i].y = -thread_data[i].y;
    }
  }

  fft_kernel_r64_b16<true>((hipFloatComplex *)thread_data, W_64);

  for(int i=0; i<32; i++) {
    int row = (lane_id/4) + (i/16)*8 + warp_id*16;
    int col = (lane_id%4) * 16 + (i%16);
    *(((float*)tile) +row*(tile_size + 2)+col) = thread_data[i].x;
  }

  for (int local_row = warp_id; local_row < tile_size; local_row+=blockDim.y) {
    int global_row = local_row + blockIdx.y * valid_tile_size;

    int local_col = lane_id;
    int global_col = local_col*2+blockIdx.x * valid_tile_size;

    if(global_row < output_size && global_col < output_size && local_row < valid_tile_size && local_col*2 < valid_tile_size) {
      *reinterpret_cast<float2 *>(d_output+global_row * output_size + global_col) = tile[local_col + (tile_size / 2 + 1) * local_row];
    }
  }

  // if(local_thread_id==0 && local_fft_id==0) {
  //   for(int i=0 ; i<tile_size;i++) {
  //     for (int j = 0; j <=tile_size/2; j++) {
  //       printf("%f %f ", tile[i*(tile_size/2+1)+j].x,
  //       tile[i*(tile_size/2+1)+j].y);
  //     }
  //     printf("\n");
  //   }
  //   printf("\n\n");
  // }
}

template <class FFT> void print_FFT_info() {
  std::cout << "FFT::storage_size: " << FFT::storage_size << std::endl;
  std::cout << "FFT::shared_memory_size: " << FFT::shared_memory_size
            << std::endl;
  std::cout << "FFT::requires_workspace: " << FFT::requires_workspace
            << std::endl;
  std::cout << "FFT::stride: " << FFT::stride << std::endl;
  DEBUG_VAR(FFT::shared_memory_size);
  DEBUG_VAR(FFT::elements_per_thread);
  std::cout << "FFT::block_dim: (" << FFT::block_dim.x << ","
            << FFT::block_dim.y << "," << FFT::block_dim.z << ")" << std::endl;
}

template <typename real_type>
void my_FFTconv(real_type *d_input, cufftdx::complex<real_type> *d_filter,
                real_type *d_output, int N, int f) {
  using complex_type = cufftdx::complex<real_type>;
  // constexpr size_t total_shared_mem =
  //     sizeof(complex_type) * tile_size * (tile_size / 2 + 1);
  // hipFuncSetAttribute(reinterpret_cast<const void*>(my_conv_kernel), hipFuncAttributeMaxDynamicSharedMemorySize,
  //                      total_shared_mem);

  hipFloatComplex h_W_64[64];
  for (int i = 0; i < 64; i++) {
    h_W_64[i] = make_hipFloatComplex(cos((-2 * M_PI * i) / 64.0),
                                    sin((-2 * M_PI * i) / 64.0));
  }

  hipFloatComplex *W_64;
  CHECK_CUDA(hipMalloc(&W_64, 64 * sizeof(hipFloatComplex)));
  CHECK_CUDA(hipMemcpy(W_64, h_W_64, 64 * sizeof(hipFloatComplex),
                        hipMemcpyHostToDevice));

  // Persisting L2 Cache
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  size_t size = std::min(int(sizeof(hipFloatComplex) * 64), prop.persistingL2CacheMaxSize);
  checkCuda(hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, size));

  hipLaunchAttributeValue stream_attribute;                                         // Stream level attributes data structure
  stream_attribute.accessPolicyWindow.base_ptr  = reinterpret_cast<void*>(W_64); // Global Memory data pointer
  stream_attribute.accessPolicyWindow.num_bytes = 64 * sizeof(hipFloatComplex);                    // Number of bytes for persistence access.
                                                                                // (Must be less than hipDeviceProp_t::accessPolicyMaxWindowSize)
  stream_attribute.accessPolicyWindow.hitRatio  = 1;                          // Hint for cache hit ratio
  stream_attribute.accessPolicyWindow.hitProp   = hipAccessPropertyPersisting; // Type of access property on cache hit
  stream_attribute.accessPolicyWindow.missProp  = hipAccessPropertyStreaming;  // Type of access property on cache miss.

  hipStream_t stream;
  checkCuda(hipStreamCreate(&stream));

  checkCuda(hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute));
  
  int tile_num = (N + tile_size - f - f + 1) / (tile_size - f + 1);
  dim3 tile_grid(tile_num, tile_num);
  GpuTimer timer;
  timer.Start();
  dim3 block_dim(32,4);
  my_conv_kernel<<<tile_grid, block_dim, 0, stream>>>(d_input, d_filter, d_output,
                                                  N, f, W_64);
  hipDeviceSynchronize();
  timer.Stop();

  // dim3 tmp={8,1};
  // shrinkage_kernel<real_type, fft_r2c, fft_c2r, fft_c2c_forward,
  //                  fft_c2c_inverse><<<1, tmp, total_shared_mem>>>(
  //     d_input, d_filter, d_output, N, f);
  checkCuda(hipDeviceSynchronize());
  checkCuda(hipGetLastError());

  float time_ms = timer.Elapsed();
  int stride = tile_size - f + 1;
  int tiles_per_dim = (N - f + stride) / stride;
  long long ops_per_tile = static_cast<long long>(
      4 * tile_size * tile_size * log2(tile_size * tile_size) +
      6 * tile_size * tile_size);
  long long total_ops =
      static_cast<long long>(tiles_per_dim) * tiles_per_dim * ops_per_tile;
  float gflops = total_ops / (time_ms * 1e6f);

  printf("[myConv] Time: %.3f ms, GFLOPS: %.2f\n", time_ms, gflops);

  // static float total_time=0;
  // static int cnt=0;
  // total_time+=time_ms;
  // cnt++;
  // printf("Avg Time: %.3f ms\n", total_time/cnt);
}

template <typename real_type>
cufftdx::complex<real_type> *preprocess_filter(real_type *h_filter, int f,
                                               int T) {
  // 1. 필터 패딩 및 wrap-around 중심 정렬
  float *padded_filter = (float *)calloc(T * T, sizeof(float));
  for (int i = 0; i < f; ++i)
    for (int j = 0; j < f; ++j)
      padded_filter[((T - i) % T) * T + (T - j) % T] =
          h_filter[i * f + j] / T / T;

  // 2. GPU 메모리 할당
  float *d_filter;
  hipfftComplex *d_filter_fft;
  hipMalloc(&d_filter, sizeof(float) * T * T);
  hipMalloc(&d_filter_fft, sizeof(hipfftComplex) * T * (T / 2 + 1));

  // 3. 복사
  hipMemcpy(d_filter, padded_filter, sizeof(float) * T * T,
             hipMemcpyHostToDevice);
  free(padded_filter);

  // 4. FFT plan 생성 및 실행
  hipfftHandle plan;
  hipfftPlan2d(&plan, T, T, HIPFFT_R2C);
  hipfftExecR2C(plan, d_filter, d_filter_fft);
  hipfftDestroy(plan);

  // 5. 중간 입력 버퍼 해제
  hipFree(d_filter);

  // 6. 결과 반환
  return reinterpret_cast<cufftdx::complex<real_type> *>(d_filter_fft);
}

void my_convolution(float *h_input, float *h_filter, float *h_output, int N,
                    int f) {
  float *d_input, *d_output;
  int out_size = N - f + 1;

  hipMalloc(&d_input, N * N * sizeof(float));
  hipMalloc(&d_output, out_size * out_size * sizeof(float));

  // Copy input data to device
  hipMemcpy(d_input, h_input, N * N * sizeof(float), hipMemcpyHostToDevice);
  auto d_filter = preprocess_filter(h_filter, f, tile_size);

  my_FFTconv<float>(d_input, d_filter, d_output, N, f);

  // Copy result back to host
  checkCuda(hipMemcpy(h_output, d_output, out_size * out_size * sizeof(float),
                       hipMemcpyDeviceToHost));

  // Free memory
  checkCuda(hipFree(d_input));
  checkCuda(hipFree(d_output));
  checkCuda(hipFree(d_filter));
}