#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "convolution.hpp"
#include "gpuTimer.h"

#define checkCuda(expr)                                                        \
  do {                                                                         \
    hipError_t err = (expr);                                                  \
    if (err != hipSuccess) {                                                  \
      printf("CUDA Error: %s\n", hipGetErrorString(err));                     \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

__global__ void pointwise_multiply(hipfftComplex *a, hipfftComplex *b, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    float ax = a[i].x, ay = a[i].y;
    float bx = b[i].x, by = b[i].y;
    a[i].x = ax * bx - ay * by;
    a[i].y = ax * by + ay * bx;
  }
}

#define CHECK_CUFFT(call)                                                      \
  do {                                                                         \
    hipfftResult err = (call);                                                  \
    if (err != HIPFFT_SUCCESS) {                                                \
      std::cerr << "[cuFFT ERROR] " << #call << " failed with code " << err    \
                << std::endl;                                                  \
      std::exit(EXIT_FAILURE);                                                 \
    }                                                                          \
  } while (0)

// assume N=2^k
void convolution_cufft_tiled(float *h_input, float *h_filter, float *h_output,
                             int N, int f, int T) {
  int out_size = N - f + 1;
  int stride = T - f + 1;
  int tiles_per_dim = (out_size + stride - 1) / stride;

  hipfftHandle planR2C, planC2R;
  checkCufft(hipfftPlan2d(&planR2C, T, T, HIPFFT_R2C), "plan r2c");
  checkCufft(hipfftPlan2d(&planC2R, T, T, HIPFFT_C2R), "plan c2r");

  float *padded_filter = (float *)calloc(T * T, sizeof(float));
  for (int i = 0; i < f; ++i)
    for (int j = 0; j < f; ++j)
      padded_filter[((T - i) % T) * T + (T - j) % T] = h_filter[i * f + j];

  float *d_input_tile, *d_filter, *d_result_tile;
  hipfftComplex *d_input_fft, *d_filter_fft;

  checkCuda(hipMalloc(&d_input_tile, sizeof(float) * T * T));
  checkCuda(hipMalloc(&d_filter, sizeof(float) * T * T));
  checkCuda(hipMalloc(&d_result_tile, sizeof(float) * T * T));
  checkCuda(hipMalloc(&d_input_fft, sizeof(hipfftComplex) * T * (T / 2 + 1)));
  checkCuda(hipMalloc(&d_filter_fft, sizeof(hipfftComplex) * T * (T / 2 + 1)));

  checkCuda(hipMemcpy(d_filter, padded_filter, sizeof(float) * T * T,
                       hipMemcpyHostToDevice));
  free(padded_filter);

  checkCufft(hipfftExecR2C(planR2C, d_filter, d_filter_fft), "exec r2c filter");

  GpuTimer timer;
  timer.Start();

  for (int ty = 0; ty < tiles_per_dim; ++ty) {
    for (int tx = 0; tx < tiles_per_dim; ++tx) {
      float *host_tile = (float *)calloc(T * T, sizeof(float));

      for (int i = 0; i < T; ++i) {
        for (int j = 0; j < T; ++j) {
          int y = ty * stride + i;
          int x = tx * stride + j;
          if (y < N && x < N) {
            host_tile[i * T + j] = h_input[y * N + x];
          }
        }
      }

      checkCuda(hipMemcpy(d_input_tile, host_tile, sizeof(float) * T * T,
                           hipMemcpyHostToDevice));
      free(host_tile);

      checkCufft(hipfftExecR2C(planR2C, d_input_tile, d_input_fft),
                 "exec r2c input");

      int n = T * (T / 2 + 1);
      pointwise_multiply<<<(n + 255) / 256, 256>>>(d_input_fft, d_filter_fft,
                                                   n);
      checkCuda(hipDeviceSynchronize());

      checkCufft(hipfftExecC2R(planC2R, d_input_fft, d_result_tile), "exec c2r");

      float *host_result = (float *)malloc(sizeof(float) * T * T);
      checkCuda(hipMemcpy(host_result, d_result_tile, sizeof(float) * T * T,
                           hipMemcpyDeviceToHost));

      for (int i = 0; i < stride && (ty * stride + i) < out_size; ++i) {
        for (int j = 0; j < stride && (tx * stride + j) < out_size; ++j) {
          h_output[(ty * stride + i) * out_size + (tx * stride + j)] =
              host_result[(i + f - 1) * T + (j + f - 1)] / (float)(T * T);
        }
      }
      free(host_result);
    }
  }

  timer.Stop();
  double log2_S2 = log2((double)T * T);
  long long ops = (long long)(tiles_per_dim * tiles_per_dim *
                              (4.0 * T * T * log2_S2 + 6.0 * T * T));
  float time_ms = timer.Elapsed();
  float gflops = ops / (time_ms * 1e6f);
  printf("[cuFFT-tiled] Time: %.3f ms, GFLOPS: %.2f\n", time_ms, gflops);

  hipFree(d_input_tile);
  hipFree(d_filter);
  hipFree(d_result_tile);
  hipFree(d_input_fft);
  hipFree(d_filter_fft);
  hipfftDestroy(planR2C);
  hipfftDestroy(planC2R);
}
