#include "hip/hip_runtime.h"
#include "gpuTimer.h"
#include <hipfft/hipfft.h>
#include <cufftdx.hpp>
#include <typeinfo>

#define DEBUG_VAR(x) std::cout << #x << ": " << (x) << std::endl;

#define checkCuda(expr)                                                        \
    do {                                                                       \
        hipError_t err = (expr);                                              \
        if (err != hipSuccess) {                                              \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,   \
                    hipGetErrorString(err));                                  \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    } while (0)

using namespace cufftdx;

static constexpr unsigned int tile_size = 64;

template <typename real_type, class fft_r2c, class fft_c2r,
          class fft_c2c_forward, class fft_c2c_inverse, class fft_single>
__global__ void conv_kernel(real_type *d_input,
                            cufftdx::complex<real_type> *d_filter,
                            real_type *d_output, int N, int f) {
    using complex_type = typename cufftdx::complex<real_type>;

    // Allocate register
    complex_type thread_data[fft_c2c_forward::storage_size];

    // Define hared memory
    // Note: must be aligned. (i.g. fft_r2c::shared_memory_size is multiple of
    // sizeof(complex)*4 = 32 -> cufftdx used aligned smem) Todo: increase tile
    // size to 256, offloading data to register
    extern __shared__ __align__(alignof(float4)) complex_type shared[];
    complex_type *shared_buffer = shared;
    __shared__ complex_type tile[tile_size * (tile_size / 2 + 1)];
    // shared + (fft_c2c_forward::shared_memory_size / sizeof(complex_type));
    // float *real_tile = reinterpret_cast<float *>(tile);
    // complex_type* tile = shared;
    // complex_type* shared_buffer = shared+tile_size*(tile_size/2+1);

    // thread idx
    const int local_thread_id = threadIdx.x;
    const int local_fft_id = threadIdx.y;

    // constant
    constexpr int fpb = fft_c2c_forward::ffts_per_block;
    constexpr int c2r_ept = fft_c2c_forward::elements_per_thread / 2;

    const int input_size = N;
    const int output_size = N - f + 1;
    const int valid_tile_size = tile_size - f + 1;

    const int input_data_bias = (blockIdx.x + blockIdx.y * N) * valid_tile_size;
    const int output_data_bias =
        (blockIdx.x + blockIdx.y * output_size) * valid_tile_size;

    // r2c
    for (int j = 0; j < (tile_size - 1) / fpb + 1; j++) {
        int local_row = local_fft_id + fpb * j;
        int global_row = local_row + blockIdx.y * valid_tile_size;

        if (local_row < tile_size) {
            auto d_input_asComplex = reinterpret_cast<complex_type *>(
                d_input + input_data_bias + local_row * input_size);
            for (int i = 0; i < (tile_size / 2 + 1 - 1) / fft_r2c::stride + 1;
                 i++) {
                int col = local_thread_id + fft_r2c::stride * i;
                if (global_row < input_size &&
                    col * 2 + blockIdx.x * valid_tile_size < input_size)
                    thread_data[i] = d_input_asComplex[col];
                else
                    thread_data[i] = complex_type(0, 0);
            }
        }
        // fft
        fft_r2c().execute(thread_data, shared_buffer);
        // // register -> smem;
        if (local_row < tile_size) {
            for (int i = 0; i < (tile_size / 2 + 1 - 1) / fft_r2c::stride + 1;
                 i++) {
                int col = local_thread_id + fft_r2c::stride * i;

                if (col < tile_size / 2 + 1)
                    tile[col + (tile_size / 2 + 1) * local_row] =
                        thread_data[i];
            }
        }
    }

    __syncthreads();
    // c2c forward & backward
    for (int j = 0; j < (tile_size / 2 + 1 - 1) / fpb + 1; j++) { // 17->16
        int col = local_fft_id + fpb * j;
        // smem -> register
        if (col < tile_size / 2 + 1) {
            for (int i = 0; i < fft_c2c_forward::elements_per_thread; i++) {
                thread_data[i] =
                    tile[(local_thread_id + fft_c2c_forward::stride * i) *
                             (tile_size / 2 + 1) +
                         col];
            }
        }

        // fft
        fft_c2c_forward().execute(thread_data, shared_buffer);

        // element wise mult
        if (col < tile_size / 2 + 1) {
            for (int i = 0; i < fft_c2c_forward::elements_per_thread; i++) {
                thread_data[i] =
                    thread_data[i] *
                    d_filter[(local_thread_id + fft_c2c_forward::stride * i) *
                                 (tile_size / 2 + 1) +
                             col];
            }
        }

        // fft
        fft_c2c_inverse().execute(thread_data, shared_buffer);

        // register -> smem
        if (col < tile_size / 2 + 1) {
            for (int i = 0; i < fft_c2c_forward::elements_per_thread; i++) {
                tile[(local_thread_id + fft_c2c_forward::stride * i) *
                         (tile_size / 2 + 1) +
                     col] = thread_data[i];
                // tile[(local_thread_id+fft_c2c_forward::stride*i)+col*tile_size]=thread_data[i];
            }
        }
    }
    __syncthreads();

    for (int j = 0; j < (valid_tile_size - 1) / fpb + 1; j++) {
        int local_row = local_fft_id + fpb * j;
        int global_row = local_row + blockIdx.y * valid_tile_size;
        // register -> smem;
        if (local_row < valid_tile_size && global_row < output_size) {
            for (int i = 0; i < (tile_size / 2 + 1 - 1) / fft_r2c::stride + 1;
                 i++) {
                int col = local_thread_id + fft_r2c::stride * i;
                if (col < tile_size / 2 + 1) {
                    thread_data[i] =
                        tile[col + (tile_size / 2 + 1) * local_row];
                    // thread_data_x[i] = tile[col*tile_size+local_row];
                }
            }
        }

        // fft
        fft_c2r().execute(thread_data, shared_buffer);

        // global mem -> register
        // if(local_row<valid_tile_size && global_row<output_size) {
        //   auto d_output_asComplex = reinterpret_cast<complex_type *>(
        //       d_output + output_data_bias + local_row * output_size);
        //   for (int i = 0, idx = local_thread_id;A idx < valid_tile_size / 2;
        //       i++, idx += fft_c2r::stride) {
        //     d_output_asComplex[idx] = thread_data[i];
        //   }
        // }

        if (local_row < valid_tile_size && global_row < output_size) {
            auto d_output_asComplex = reinterpret_cast<complex_type *>(
                d_output + output_data_bias + local_row * output_size);
            for (int i = 0; i < (tile_size / 2 + 1 - 1) / fft_r2c::stride + 1;
                 i++) {
                int local_col = local_thread_id + fft_r2c::stride * i;
                int global_col = local_col * 2 + blockIdx.x * valid_tile_size;
                if (local_col < valid_tile_size / 2 &&
                    global_col < output_size) {
                    d_output_asComplex[local_col] = thread_data[i];
                    // tile[local_col+(tile_size/2+1)*local_row] =
                    // thread_data[i];
                }
            }
        }
    }
}

template <class FFT> void print_FFT_info() {
    std::cout << "FFT::storage_size: " << FFT::storage_size << std::endl;
    std::cout << "FFT::shared_memory_size: " << FFT::shared_memory_size
              << std::endl;
    std::cout << "FFT::requires_workspace: " << FFT::requires_workspace
              << std::endl;
    std::cout << "FFT::stride: " << FFT::stride << std::endl;
    DEBUG_VAR(FFT::shared_memory_size);
    DEBUG_VAR(FFT::elements_per_thread);
    std::cout << "FFT::block_dim: (" << FFT::block_dim.x << ","
              << FFT::block_dim.y << "," << FFT::block_dim.z << ")"
              << std::endl;
}

template <typename real_type>
void FFTconv(real_type *d_input, cufftdx::complex<real_type> *d_filter,
             real_type *d_output, int N, int f) {
    using complex_type = cufftdx::complex<real_type>;
    static constexpr unsigned int Arch = CUFFT_TARGET_ARCHS;

    // Kernel settings
    static constexpr unsigned int ept = 4;  // element per thread
    static constexpr unsigned int fpb = 16; // fft per block

    using fft_base = decltype(Block() + Precision<real_type>() + SM<Arch>() +
                              Size<tile_size>() + ElementsPerThread<ept>() +
                              FFTsPerBlock<fpb>());

    using fft_r2c =
        decltype(fft_base() + Type<fft_type::r2c>() +
                 RealFFTOptions<complex_layout::natural, real_mode::folded>());
    using fft_c2r =
        decltype(fft_base() + Type<fft_type::c2r>() +
                 RealFFTOptions<complex_layout::natural, real_mode::folded>());
    using fft_c2c_forward = decltype(fft_base() + Type<fft_type::c2c>() +
                                     Direction<fft_direction::forward>());
    using fft_c2c_inverse = decltype(fft_base() + Type<fft_type::c2c>() +
                                     Direction<fft_direction::inverse>());

    using fft_single = decltype(
        Block() + Precision<real_type>() + SM<Arch>() + Size<tile_size>() +
        ElementsPerThread<ept / fpb>() + FFTsPerBlock<1>() +
        Type<fft_type::c2c>() + Direction<fft_direction::forward>());

    // std::cout << "fft_r2c" << std::endl; print_FFT_info<fft_r2c>();
    // std::cout << "fft_c2r" << std::endl; print_FFT_info<fft_c2r>();
    // std::cout << "fft_c2c_forward" << std::endl;
    // print_FFT_info<fft_c2c_forward>(); std::cout << "fft_c2c_backward" <<
    // std::endl; print_FFT_info<fft_c2c_inverse>(); std::cout << "fft_single"
    // << std::endl; print_FFT_info<fft_single>();

    constexpr size_t total_shared_mem = fft_c2c_forward::shared_memory_size;
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        conv_kernel<real_type), fft_r2c, fft_c2r, fft_c2c_forward,
                    fft_c2c_inverse, fft_single>,
        hipFuncAttributeMaxDynamicSharedMemorySize, total_shared_mem);

    int tile_num = (N + tile_size - f - f + 1) / (tile_size - f + 1);
    dim3 tile_grid(tile_num, tile_num);
    hipStream_t stream;
    checkCuda(hipStreamCreate(&stream));
    GpuTimer timer(stream);
    timer.Start();
    // shrinkage_kernel<real_type, fft_r2c, fft_c2r, fft_c2c_forward,
    //                   fft_c2c_inverse,fft_single><<<tile_num*tile_num,
    //                   fft_r2c::block_dim, total_shared_mem>>>(
    //     d_input, d_filter, d_output, N, f);
    for (int i = 0; i < 1000; i++) {
        conv_kernel<real_type, fft_r2c, fft_c2r, fft_c2c_forward,
                    fft_c2c_inverse, fft_single>
            <<<tile_grid, fft_r2c::block_dim, total_shared_mem, stream>>>(
                d_input, d_filter, d_output, N, f);
    }
    timer.Stop();
    checkCuda(hipGetLastError());
    checkCuda(hipStreamDestroy(stream));

    float time_ms = timer.Elapsed();
    int stride = tile_size - f + 1;
    int tiles_per_dim = (N - f + stride) / stride;
    long long ops_per_tile = static_cast<long long>(
        4 * tile_size * tile_size * log2(tile_size * tile_size) +
        6 * tile_size * tile_size);
    long long total_ops =
        static_cast<long long>(tiles_per_dim) * tiles_per_dim * ops_per_tile;
    float gflops = total_ops / (time_ms * 1e6f);

    printf("[cuFFTDx] Time: %.3f ms, GFLOPS: %.2f\n", time_ms / 1000, gflops * 1000);

    // static float total_time=0;
    // static int cnt=0;
    // total_time+=time_ms;
    // cnt++;
    // printf("Avg Time: %.3f ms\n", total_time/cnt);
}

template <typename real_type>
cufftdx::complex<real_type> *preprocess_filter(real_type *h_filter, int f,
                                               int T) {
    // 1. 필터 패딩 및 wrap-around 중심 정렬
    float *padded_filter = (float *)calloc(T * T, sizeof(float));
    for (int i = 0; i < f; ++i)
        for (int j = 0; j < f; ++j)
            padded_filter[((T - i) % T) * T + (T - j) % T] =
                h_filter[i * f + j] / T / T;

    // 2. GPU 메모리 할당
    float *d_filter;
    hipfftComplex *d_filter_fft;
    hipMalloc(&d_filter, sizeof(float) * T * T);
    hipMalloc(&d_filter_fft, sizeof(hipfftComplex) * T * (T / 2 + 1));

    // 3. 복사
    hipMemcpy(d_filter, padded_filter, sizeof(float) * T * T,
               hipMemcpyHostToDevice);
    free(padded_filter);

    // 4. FFT plan 생성 및 실행
    hipfftHandle plan;
    hipfftPlan2d(&plan, T, T, HIPFFT_R2C);
    hipfftExecR2C(plan, d_filter, d_filter_fft);
    hipfftDestroy(plan);

    // 5. 중간 입력 버퍼 해제
    hipFree(d_filter);

    // 6. 결과 반환
    return reinterpret_cast<cufftdx::complex<real_type> *>(d_filter_fft);
}

void convolution_cufftdx(float *h_input, float *h_filter, float *h_output,
                         int N, int f) {
    float *d_input, *d_output;
    int out_size = N - f + 1;

    hipMalloc(&d_input, N * N * sizeof(float));
    hipMalloc(&d_output, out_size * out_size * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, h_input, N * N * sizeof(float), hipMemcpyHostToDevice);
    auto d_filter = preprocess_filter(h_filter, f, tile_size);

    FFTconv<float>(d_input, d_filter, d_output, N, f);

    // Copy result back to host
    checkCuda(hipMemcpy(h_output, d_output,
                         out_size * out_size * sizeof(float),
                         hipMemcpyDeviceToHost));

    // Free memory
    checkCuda(hipFree(d_input));
    checkCuda(hipFree(d_output));
    checkCuda(hipFree(d_filter));
}