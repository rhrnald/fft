#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>

#include "helper.h"
#include "stat.h"

#define CHECK_CUDA(call)                                                       \
    do {                                                                       \
        hipError_t err = call;                                                \
        if (err != hipSuccess) {                                              \
            fprintf(stderr, "CUDA error in %s (%s:%d): %s\n", #call, __FILE__, \
                    __LINE__, hipGetErrorString(err));                        \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    } while (0)

#define CHECK_CUFFT(call)                                                      \
    do {                                                                       \
        hipfftResult err = call;                                                \
        if (err != HIPFFT_SUCCESS) {                                            \
            fprintf(stderr, "cuFFT error in %s (%s:%d): %d\n", #call,          \
                    __FILE__, __LINE__, err);                                  \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    } while (0)

void baseline_fft(float2 *h_input, float2 *h_output, int N, int batch) {
    printf("running baseline (type=float, N=%d, batch=%d)\n", N, batch);
    static constexpr unsigned int kernel_runs = 10;
    static constexpr unsigned int warm_up_runs = 1;

    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));

    float2 *d_input, *d_output;
    hipMalloc(&d_input, sizeof(float2) * N * batch);
    hipMalloc(&d_output, sizeof(float2) * N * batch);
    hipMemcpy(d_input, h_input, sizeof(float2) * N * batch,
               hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan1d(&plan, N, HIPFFT_C2C, batch);

    double elapsedTime = measure_execution_ms(
        [&](hipStream_t stream) {
            hipfftExecC2C(plan, (hipfftComplex *)d_input,
                         (hipfftComplex *)d_output, HIPFFT_FORWARD);
            // assert("4096 half is not supported" && false);
        },
        warm_up_runs, kernel_runs, stream);

    hipMemcpy(h_output, d_output, sizeof(float2) * N * batch,
               hipMemcpyDeviceToHost);

    stat::push(stat::RunStat{
        /*type*/ "baseline", // 자유롭게 "cufft" 등으로 바꿔도 됨
        /*N*/ static_cast<unsigned>(N),
        /*radix*/ 0, // baseline이라 없음
        /*B*/ static_cast<unsigned>(batch),
        /*max_err*/ 0.0,
        /*comp_ms*/ 0.0,
        /*comm_ms*/ 0.0,
        /*e2e_ms*/ elapsedTime
    });

    hipFree(d_input);
    hipFree(d_output);
    hipfftDestroy(plan);
}
