#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include "my_fft.h"
#include "fft_tc_sm.h"

void baseline_fft(float2 *d_data, int len, int batch, int N);

void check_result(const float2* ref, const half2* test, int N,
                  float atol = 1e-2f, float rtol = 1e-2f) {
    float max_abs_err = 0.0f;
    float max_rel_err = 0.0f;
    int   max_idx_abs = -1;
    int   max_idx_rel = -1;

    int bad_cnt = 0;

    for (int i = 0; i < N; ++i) {
        // half2 -> float2 변환 (정식 방법)
        float2 tf = __half22float2(test[i]);  // tf.x, tf.y 가 float

        // 차이 계산
        float dx = ref[i].x - tf.x;
        float dy = ref[i].y - tf.y;
        float abs_err = std::sqrt(dx * dx + dy * dy);

        float ref_mag = std::sqrt(ref[i].x * ref[i].x + ref[i].y * ref[i].y);
        float rel_err = abs_err / (ref_mag + 1e-20f);

        if (abs_err > max_abs_err) { max_abs_err = abs_err; max_idx_abs = i; }
        if (rel_err > max_rel_err) { max_rel_err = rel_err; max_idx_rel = i; }

        // 허용 오차 밖이면 몇 개만 샘플로 출력
        bool fail = abs_err > (atol + rtol * ref_mag);
        if (fail && bad_cnt < 10) {
            printf("mismatch @%d: ref=(%.7f, %.7f) test=(%.7f, %.7f) "
                        "abs_err=%.7g rel_err=%.7g\n",
                        i, ref[i].x, ref[i].y, tf.x, tf.y, abs_err, rel_err);
            ++bad_cnt;
        }
    }

    printf("Max abs err = %.7g at i=%d\n", max_abs_err, max_idx_abs);
    printf("Max rel err = %.7g at i=%d\n", max_rel_err, max_idx_rel);

    if (bad_cnt == 0) {
        printf("All %d elements within tolerance (atol=%.2e, rtol=%.2e)\n", N, atol, rtol);
    } else {
        printf("%d elements exceeded tolerance (atol=%.2e, rtol=%.2e)\n", bad_cnt, atol, rtol);
    }
}

template <typename T>
float2 to_float2(const T& v);

template <>
float2 to_float2<float2>(const float2& v) { return v; }

template <>
float2 to_float2<half2>(const half2& v) { return __half22float2(v); }

template <typename T>
float check_max_abs_err(const float2* ref, const T* test, int N) {
    float max_abs_err = 0.0f;

    #pragma unroll
    for (int i = 0; i < N; ++i) {
        float2 tf = to_float2<T>(test[i]);
        float dx = ref[i].x - tf.x;
        float dy = ref[i].y - tf.y;
        float abs_err = sqrtf(dx * dx + dy * dy);
        if (abs_err > max_abs_err) max_abs_err = abs_err;
    }

    // 요구사항: "maximum absolute error만 출력"
    return max_abs_err;
}

int main() {
    constexpr long long batch = 65536;
    constexpr long long len = 64;
    constexpr long long N = batch * len;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    printf("Max grid size: x=%d, y=%d, z=%d\n", prop.maxGridSize[0],
           prop.maxGridSize[1], prop.maxGridSize[2]);

    hipFloatComplex *h_input = (hipFloatComplex *)malloc(sizeof(hipFloatComplex) * N);
    half2 *h_input_half = (half2 *)malloc(sizeof(half2) * N);
    for (int i = 0; i < N; ++i) {
        h_input[i].x = sinf(2 * M_PI * (i%len) / 64); // real part
        h_input[i].y = 0.0f;                   // imag part
        // h_input[i].x = i % len;
        // h_input[i].y = 0;
        h_input_half[i] = make_half2(h_input[i].x, h_input[i].y);
    }

    hipFloatComplex *d_baseline;
    hipFloatComplex *d_custom;
    half2 *d_custom_half;

    hipMalloc(&d_baseline, sizeof(hipFloatComplex) * N);
    hipMalloc(&d_custom, sizeof(hipFloatComplex) * N);
    hipMalloc(&d_custom_half, sizeof(half2) * N);

    hipMemcpy(d_baseline, h_input, sizeof(hipFloatComplex) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_custom, h_input, sizeof(hipFloatComplex) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_custom_half, h_input_half, sizeof(half2) * N,
               hipMemcpyHostToDevice);

    baseline_fft(d_baseline, len, batch, N);
    // my_fft<float2, N>(d_baseline);

    
    fft_tc_sm_val<half, len, 8>(d_custom_half, batch);
    fft_tc_sm_val<float, len, 8>(d_custom, batch);


    hipFloatComplex *h_baseline = (hipFloatComplex *)malloc(sizeof(hipFloatComplex) * N);
    hipFloatComplex *h_custom = (hipFloatComplex *)malloc(sizeof(hipFloatComplex) * N);
    half2 *h_custom_half = (half2 *)malloc(sizeof(half2) * N);

    hipMemcpy(h_baseline, d_baseline, sizeof(hipFloatComplex) * N,
               hipMemcpyDeviceToHost);
    hipMemcpy(h_custom, d_custom, sizeof(hipFloatComplex) * N,
               hipMemcpyDeviceToHost);
    hipMemcpy(h_custom_half, d_custom_half, sizeof(half2) * N,
               hipMemcpyDeviceToHost);

    printf("[val] half N=64 radix=8 max abs err = %.7g\n", check_max_abs_err(h_baseline, h_custom_half, N));
    printf("[val] float N=64 radix=8 max abs err = %.7g\n", check_max_abs_err(h_baseline, h_custom, N));

    fft_tc_sm_perf<half, len, 8>(d_custom_half, batch);
    fft_tc_sm_perf<float, len, 8>(d_custom, batch);

    free(h_input);
    free(h_baseline);
    free(h_custom);
    hipFree(d_baseline);
    hipFree(d_custom);
    return 0;
}
