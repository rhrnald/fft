#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include "stat.h"
#include "utils.h"

#include "fft_tc_sm_bench.h"

void baseline_fft(float2 *h_input, float2 *h_output, int N, int batch);

int main() {
    constexpr long long N = 64;
    constexpr long long batch = 65536;

    float2 *h_input = (float2 *)malloc(sizeof(float2) * N * batch);
    half2 *h_input_half = (half2 *)malloc(sizeof(half2) * N * batch);
    float2 *h_output = (float2 *)malloc(sizeof(float2) * N * batch);

    for (int i = 0; i < N; ++i) {
        h_input[i].x = sinf(2 * M_PI * (i % N) / 64);
        // h_input[i].x = i % N;
        h_input[i].y = 0.0f;
        h_input_half[i] = make_half2(h_input[i].x, h_input[i].y);
    }

    baseline_fft(h_input, h_output, N, batch);

    // my_fft<float2, N>(d_custom_half);

    fft_tc_sm_benchmark<N>(h_input, h_input_half, h_output, batch);

    stat::set_title("FFT benchmark results");
    stat::print_table();

    return 0;
}
