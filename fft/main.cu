#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include "stat.h"
#include "utils.h"

#include "my_fft.h"
#include "fft_tc_sm_bench.h"
#include "my_fft.h"


void baseline_fft(float2 *h_input, float2 *h_output, int N, int batch);


int main() {
    constexpr long long N = 64;
    constexpr long long batch = 65536;
    
    float2 *h_input = (float2 *)malloc(sizeof(float2) * N * batch);
    half2 *h_input_half = (half2 *)malloc(sizeof(half2) * N * batch);
    float2 *h_answer = (float2 *)malloc(sizeof(float2) * N * batch);

    for (int i = 0; i < N * batch; ++i) {
        h_input[i].x = sinf(2 * M_PI * (i % N) / 64);
        // h_input[i].x = i % N;
        h_input[i].y = 0.0f;

        h_input_half[i] = make_half2(h_input[i].x, h_input[i].y);
    }

    baseline_fft(h_input, h_answer, N, batch);

    float2 *d_input;
    CHECK_CUDA(hipMalloc(&d_input, sizeof(float2) * N * batch));
    CHECK_CUDA(hipMemcpy(d_input, h_input, sizeof(float2) * N * batch, hipMemcpyHostToDevice));

    my_fft_benchmark<N>(h_input, h_input_half, h_answer, batch);
    // stat::print_table();

    fft_tc_sm_benchmark<N>(h_input, h_input_half, h_answer, batch);
    // stat::print_table();

    // fft_tc_sm_benchmark<256>(h_input, h_input_half, answer, batch);
    

    
    stat::set_title("FFT benchmark results");
    stat::print_table();
    return 0;
}
