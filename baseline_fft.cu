#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK_CUDA(call)                                                       \
do {                                                                           \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
        fprintf(stderr, "CUDA error in %s (%s:%d): %s\n",                      \
                #call, __FILE__, __LINE__, hipGetErrorString(err));          \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define CHECK_CUFFT(call)                                                      \
do {                                                                           \
    hipfftResult err = call;                                                    \
    if (err != HIPFFT_SUCCESS) {                                                \
        fprintf(stderr, "cuFFT error in %s (%s:%d): %d\n",                     \
                #call, __FILE__, __LINE__, err);                               \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

void baseline_fft(float2* d_data, int N) {
    hipfftHandle plan;
    hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, (hipfftComplex*)d_data, (hipfftComplex*)d_data, HIPFFT_FORWARD);
    // cufftExecC2C(plan, (cufftComplex*)d_data, (cufftComplex*)d_data, CUFFT_INVERSE);
    hipfftDestroy(plan);
}

