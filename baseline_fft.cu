#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK_CUDA(call)                                                       \
do {                                                                           \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
        fprintf(stderr, "CUDA error in %s (%s:%d): %s\n",                      \
                #call, __FILE__, __LINE__, hipGetErrorString(err));          \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define CHECK_CUFFT(call)                                                      \
do {                                                                           \
    hipfftResult err = call;                                                    \
    if (err != HIPFFT_SUCCESS) {                                                \
        fprintf(stderr, "cuFFT error in %s (%s:%d): %d\n",                     \
                #call, __FILE__, __LINE__, err);                               \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

void baseline_fft(float2* d_data, int N) {
    hipfftHandle plan;
    hipfftPlan1d(&plan, 64, HIPFFT_C2C, N/64);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    hipfftExecC2C(plan, (hipfftComplex*)d_data, (hipfftComplex*)d_data, HIPFFT_FORWARD);
    hipEventRecord(stop);
    hipDeviceSynchronize();
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Baseline FFT Time taken: %f ms\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    // cufftExecC2C(plan, (cufftComplex*)d_data, (cufftComplex*)d_data, CUFFT_INVERSE);
    hipfftDestroy(plan);
}

